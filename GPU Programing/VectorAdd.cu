﻿ 
#include <hip/hip_runtime.h> 
#include <stdlib.h> 
#include <stdio.h>

__global__ void VectorAdd(int *a, int *b, int *c, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	c[tid] = a[tid] + b[tid];
}

int main() {
	const int size = 512 * 65535;
	const int BufferSize = size * sizeof(int);

	int* a;
	int* b;
	int* c;

	a = (int*)malloc(BufferSize);
	b = (int*)malloc(BufferSize);
	c = (int*)malloc(BufferSize);

	int i = 0;

	for (i = 0; i < size; i++) {
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	} 

	int* d_a;
	int* d_b;
	int* d_c;

	hipMalloc((void**)&d_a, BufferSize);
	hipMalloc((void**)&d_b, BufferSize);
	hipMalloc((void**)&d_c, BufferSize);

	hipMemcpy(d_a, a, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, BufferSize, hipMemcpyHostToDevice);

	VectorAdd <<<65535, 512 >>> (d_a, d_b, d_c, size);

	hipMemcpy(c, d_c, BufferSize, hipMemcpyDeviceToHost);

	for (i = 0; i < 5; i++) {
		printf(" Result[%d] : %d\n", i, c[i]);
	}
	printf("......\n");
	for (i = size - 5; i < size; i++) {
		printf(" Result[%d] : %d\n", i, c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(a);
	free(b);
	free(c);

	return 0;
}